// In this assignment you will write a basic kernel where every thread 
// will write out to console string "Hello world!".
// You will also initialize GPU using cudaSetDevice() and also launch
// your "Hello world" kernel.

#include <stdio.h>
#include <stdlib.h>
// we have to include few more things
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

//----------------------------------------------------------------------
// TASK 2: Write a "Hello world" kernel
// 
// Remember that kernel is defined by __global__ and inside it looks like
// a serial code for CPU. For printing out to console you can use printf().

// write your kernel here

//----------------------------------------------------------------------
__global__ void hello_world_kernel(float *x) {
  printf("Hello World\n");
}

int main(void) {
  //----------------------------------------------------------------------
  // TASK 1: Initiate GPU using cudaSetDevice()
  //
  // You can also try to write a check if there is a device with that id,
  // so the code behaves nicely when it fails
  
  // write you GPU initialization here

  //----------------------------------------------------------------------
  int deviceid = 0;
  int devCount;
  hipGetDeviceCount(&devCount);
  if (deviceid < devCount) hipSetDevice(deviceid);
  else return 1;

  
  
  //----------------------------------------------------------------------
  // TASK 3: execute your "Hello world" kernel on 1 block with 5 threads 
  //         using execution configuration syntax.
  // 
  // You may use whatever syntax version you prefer, a simplified one 
  // dimensional or full three dimensional call using dim3 data type.
  
  // execute your "Hello world" kernel here
  
  //---------------------------------------------------------------------
  float *h_x, *d_x;
  int nblocks = 1, nthreads = 5, nsize = 1 * 5;
  h_x = (float *) malloc(nsize * sizeof(float));
  hipMalloc((void **) &d_x, nsize * sizeof(float));
  hello_world_kernel<<<nblocks, nthreads>>>(d_x);
  hipMemcpy(h_x, d_x, nsize * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_x);
  free(h_x);
 
  hipDeviceReset(); 
  return (0);
}
